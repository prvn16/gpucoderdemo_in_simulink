#include "hip/hip_runtime.h"
/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 * File: sobelEdge.cu
 *
 * GPU Coder version                    : 1.0
 * CUDA/C/C++ source code generated on  : 14-Jan-2018 07:15:05
 */

/* Include Files */
#include "rt_nonfinite.h"
#include "sobelEdge.h"

/* Variable Definitions */
__constant__ int8_T const_b[9];
__constant__ int8_T b_const_b[9];

/* Function Declarations */
static __global__ void sobelEdge_kernel1(real32_T *expanded);
static __global__ void sobelEdge_kernel2(const real32_T *Image, real32_T
  *expanded);
static __global__ void sobelEdge_kernel3(real32_T *expanded, real32_T *resX);
static __global__ void sobelEdge_kernel4(real32_T *expanded);
static __global__ void sobelEdge_kernel5(const real32_T *Image, real32_T
  *expanded);
static __global__ void sobelEdge_kernel6(real32_T *expanded, real32_T *resY);
static __global__ void sobelEdge_kernel7(real32_T *resX, real32_T *magnitude);
static __global__ void sobelEdge_kernel8(real32_T *resY, real32_T *magnitude,
  real32_T *resX);

/* Function Definitions */

/*
 * Arguments    : uint3 blockArg
 *                uint3 gridArg
 *                real32_T *expanded
 * Return Type  : void
 */
static __global__ __launch_bounds__(512, 1) void sobelEdge_kernel1(real32_T
  *expanded)
{
  int32_T k;
  ;
  ;
  k = (int32_T)(((((gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y)
                   + blockIdx.x) * (blockDim.x * blockDim.y * blockDim.z) +
                  threadIdx.z * blockDim.x * blockDim.y) + threadIdx.y *
                 blockDim.x) + threadIdx.x);
  if (!(k >= 77924)) {
    expanded[k] = 0.0F;
  }
}

/*
 * Arguments    : uint3 blockArg
 *                uint3 gridArg
 *                const real32_T *Image
 *                real32_T *expanded
 * Return Type  : void
 */
static __global__ __launch_bounds__(512, 1) void sobelEdge_kernel2(const
  real32_T *Image, real32_T *expanded)
{
  uint32_T threadId;
  int32_T k;
  int32_T ocol;
  ;
  ;
  threadId = ((((gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y) +
                blockIdx.x) * (blockDim.x * blockDim.y * blockDim.z) +
               threadIdx.z * blockDim.x * blockDim.y) + threadIdx.y * blockDim.x)
    + threadIdx.x;
  ocol = (int32_T)(threadId / 240U);
  k = (int32_T)(threadId - (uint32_T)ocol * 240U);
  if ((!(k >= 240)) && (!(ocol >= 320))) {
    expanded[(k + 242 * (1 + ocol)) + 1] = Image[k + 240 * ocol];
  }
}

/*
 * Arguments    : uint3 blockArg
 *                uint3 gridArg
 *                real32_T *expanded
 *                real32_T *resX
 * Return Type  : void
 */
static __global__ __launch_bounds__(1024, 1) void sobelEdge_kernel3(real32_T
  *expanded, real32_T *resX)
{
  real32_T cv;
  int32_T n;
  int32_T k;
  int32_T threadIdY;
  int32_T threadIdX;
  __shared__ real32_T expanded_shared[1156];
  int32_T baseR;
  int32_T srow;
  int32_T strideRow;
  int32_T scol;
  int32_T strideCol;
  int32_T y_idx;
  int32_T baseC;
  int32_T x_idx;
  ;
  ;
  threadIdY = (int32_T)(blockDim.y * blockIdx.y + threadIdx.y);
  threadIdX = (int32_T)(blockDim.x * blockIdx.x + threadIdx.x);
  baseR = threadIdX;
  srow = (int32_T)threadIdx.x;
  strideRow = (int32_T)blockDim.x;
  scol = (int32_T)threadIdx.y;
  strideCol = (int32_T)blockDim.y;
  for (y_idx = srow; y_idx <= 33; y_idx += strideRow) {
    baseC = threadIdY;
    for (x_idx = scol; x_idx <= 33; x_idx += strideCol) {
      if ((baseR >= 0) && (baseR < 242) && ((baseC >= 0) && (baseC < 322))) {
        expanded_shared[y_idx + 34 * x_idx] = (real32_T)expanded[242 * baseC +
          baseR];
      } else {
        expanded_shared[y_idx + 34 * x_idx] = 0.0F;
      }

      baseC += strideCol;
    }

    baseR += strideRow;
  }

  __syncthreads();
  if ((!(threadIdX >= 240)) && (!(threadIdY >= 320))) {
    cv = 0.0F;
    for (n = 0; n < 3; n++) {
      for (k = 0; k < 3; k++) {
        cv += expanded_shared[((int32_T)threadIdx.x + ((k + threadIdX) -
          threadIdX)) + 34 * ((int32_T)threadIdx.y + ((n + threadIdY) -
          threadIdY))] * (real32_T)const_b[(3 * (2 - n) - k) + 2];
      }
    }

    resX[threadIdX + 240 * threadIdY] = cv;
  }
}

/*
 * Arguments    : uint3 blockArg
 *                uint3 gridArg
 *                real32_T *expanded
 * Return Type  : void
 */
static __global__ __launch_bounds__(512, 1) void sobelEdge_kernel4(real32_T
  *expanded)
{
  int32_T k;
  ;
  ;
  k = (int32_T)(((((gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y)
                   + blockIdx.x) * (blockDim.x * blockDim.y * blockDim.z) +
                  threadIdx.z * blockDim.x * blockDim.y) + threadIdx.y *
                 blockDim.x) + threadIdx.x);
  if (!(k >= 77924)) {
    expanded[k] = 0.0F;
  }
}

/*
 * Arguments    : uint3 blockArg
 *                uint3 gridArg
 *                const real32_T *Image
 *                real32_T *expanded
 * Return Type  : void
 */
static __global__ __launch_bounds__(512, 1) void sobelEdge_kernel5(const
  real32_T *Image, real32_T *expanded)
{
  uint32_T threadId;
  int32_T k;
  int32_T ocol;
  ;
  ;
  threadId = ((((gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y) +
                blockIdx.x) * (blockDim.x * blockDim.y * blockDim.z) +
               threadIdx.z * blockDim.x * blockDim.y) + threadIdx.y * blockDim.x)
    + threadIdx.x;
  ocol = (int32_T)(threadId / 240U);
  k = (int32_T)(threadId - (uint32_T)ocol * 240U);
  if ((!(k >= 240)) && (!(ocol >= 320))) {
    expanded[(k + 242 * (1 + ocol)) + 1] = Image[k + 240 * ocol];
  }
}

/*
 * Arguments    : uint3 blockArg
 *                uint3 gridArg
 *                real32_T *expanded
 *                real32_T *resY
 * Return Type  : void
 */
static __global__ __launch_bounds__(1024, 1) void sobelEdge_kernel6(real32_T
  *expanded, real32_T *resY)
{
  real32_T cv;
  int32_T n;
  int32_T k;
  int32_T threadIdY;
  int32_T threadIdX;
  __shared__ real32_T expanded_shared[1156];
  int32_T baseR;
  int32_T srow;
  int32_T strideRow;
  int32_T scol;
  int32_T strideCol;
  int32_T y_idx;
  int32_T baseC;
  int32_T x_idx;
  ;
  ;
  threadIdY = (int32_T)(blockDim.y * blockIdx.y + threadIdx.y);
  threadIdX = (int32_T)(blockDim.x * blockIdx.x + threadIdx.x);
  baseR = threadIdX;
  srow = (int32_T)threadIdx.x;
  strideRow = (int32_T)blockDim.x;
  scol = (int32_T)threadIdx.y;
  strideCol = (int32_T)blockDim.y;
  for (y_idx = srow; y_idx <= 33; y_idx += strideRow) {
    baseC = threadIdY;
    for (x_idx = scol; x_idx <= 33; x_idx += strideCol) {
      if ((baseR >= 0) && (baseR < 242) && ((baseC >= 0) && (baseC < 322))) {
        expanded_shared[y_idx + 34 * x_idx] = (real32_T)expanded[242 * baseC +
          baseR];
      } else {
        expanded_shared[y_idx + 34 * x_idx] = 0.0F;
      }

      baseC += strideCol;
    }

    baseR += strideRow;
  }

  __syncthreads();
  if ((!(threadIdX >= 240)) && (!(threadIdY >= 320))) {
    cv = 0.0F;
    for (n = 0; n < 3; n++) {
      for (k = 0; k < 3; k++) {
        cv += expanded_shared[((int32_T)threadIdx.x + ((k + threadIdX) -
          threadIdX)) + 34 * ((int32_T)threadIdx.y + ((n + threadIdY) -
          threadIdY))] * (real32_T)b_const_b[(3 * (2 - n) - k) + 2];
      }
    }

    resY[threadIdX + 240 * threadIdY] = cv;
  }
}

/*
 * Arguments    : uint3 blockArg
 *                uint3 gridArg
 *                real32_T *resX
 *                real32_T *magnitude
 * Return Type  : void
 */
static __global__ __launch_bounds__(512, 1) void sobelEdge_kernel7(real32_T
  *resX, real32_T *magnitude)
{
  int32_T k;
  ;
  ;
  k = (int32_T)(((((gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y)
                   + blockIdx.x) * (blockDim.x * blockDim.y * blockDim.z) +
                  threadIdx.z * blockDim.x * blockDim.y) + threadIdx.y *
                 blockDim.x) + threadIdx.x);
  if (!(k >= 76800)) {
    magnitude[k] = resX[k] * resX[k];
  }
}

/*
 * Arguments    : uint3 blockArg
 *                uint3 gridArg
 *                real32_T *resY
 *                real32_T *magnitude
 *                real32_T *resX
 * Return Type  : void
 */
static __global__ __launch_bounds__(512, 1) void sobelEdge_kernel8(real32_T
  *resY, real32_T *magnitude, real32_T *resX)
{
  int32_T k;
  ;
  ;
  k = (int32_T)(((((gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y)
                   + blockIdx.x) * (blockDim.x * blockDim.y * blockDim.z) +
                  threadIdx.z * blockDim.x * blockDim.y) + threadIdx.y *
                 blockDim.x) + threadIdx.x);
  if (!(k >= 76800)) {
    resX[k] = resY[k] * resY[k];
    magnitude[k] += resX[k];
    magnitude[k] = sqrtf(magnitude[k]);
    if ((real_T)magnitude[k] < 0.4) {
      magnitude[k] = 0.0F;
    }
  }
}

/*
 * Copyright 2017 The MathWorks, Inc.
 * Arguments    : const real32_T Image[76800]
 *                real32_T magnitude[76800]
 * Return Type  : void
 */
void sobelEdge(const real32_T Image[76800], real32_T magnitude[76800])
{
  static const int8_T b[9] = { -1, -2, -1, 0, 0, 0, 1, 2, 1 };

  static const int8_T b_b[9] = { -1, 0, 1, -2, 0, 2, -1, 0, 1 };

  real32_T *gpu_expanded;
  real32_T *gpu_Image;
  real32_T *gpu_resX;
  real32_T *gpu_resY;
  real32_T *gpu_magnitude;
  hipMalloc(&gpu_magnitude, 307200ULL);
  hipMalloc(&gpu_resY, 307200ULL);
  hipMalloc(&gpu_resX, 307200ULL);
  hipMalloc(&gpu_Image, 307200ULL);
  hipMalloc(&gpu_expanded, 311696ULL);
  sobelEdge_kernel1<<<dim3(153U, 1U, 1U), dim3(512U, 1U, 1U)>>>(gpu_expanded);
  hipMemcpy((void *)gpu_Image, (void *)&Image[0], 307200ULL,
             hipMemcpyHostToDevice);
  sobelEdge_kernel2<<<dim3(150U, 1U, 1U), dim3(512U, 1U, 1U)>>>(gpu_Image,
    gpu_expanded);
  hipMemcpyToSymbol(HIP_SYMBOL(const_b), b, 9ULL, 0ULL, hipMemcpyHostToDevice);
  sobelEdge_kernel3<<<dim3(8U, 10U, 1U), dim3(32U, 32U, 1U)>>>(gpu_expanded,
    gpu_resX);
  sobelEdge_kernel4<<<dim3(153U, 1U, 1U), dim3(512U, 1U, 1U)>>>(gpu_expanded);
  sobelEdge_kernel5<<<dim3(150U, 1U, 1U), dim3(512U, 1U, 1U)>>>(gpu_Image,
    gpu_expanded);
  hipMemcpyToSymbol(HIP_SYMBOL(b_const_b), b_b, 9ULL, 0ULL, hipMemcpyHostToDevice);
  sobelEdge_kernel6<<<dim3(8U, 10U, 1U), dim3(32U, 32U, 1U)>>>(gpu_expanded,
    gpu_resY);
  sobelEdge_kernel7<<<dim3(150U, 1U, 1U), dim3(512U, 1U, 1U)>>>(gpu_resX,
    gpu_magnitude);
  sobelEdge_kernel8<<<dim3(150U, 1U, 1U), dim3(512U, 1U, 1U)>>>(gpu_resY,
    gpu_magnitude, gpu_resX);
  hipMemcpy((void *)&magnitude[0], (void *)gpu_magnitude, 307200ULL,
             hipMemcpyDeviceToHost);
  hipFree(gpu_expanded);
  hipFree(gpu_Image);
  hipFree(gpu_resX);
  hipFree(gpu_resY);
  hipFree(gpu_magnitude);
}

/*
 * File trailer for sobelEdge.cu
 *
 * [EOF]
 */
