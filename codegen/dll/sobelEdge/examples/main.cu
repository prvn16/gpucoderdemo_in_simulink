/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 * File: main.cu
 *
 * GPU Coder version                    : 1.0
 * CUDA/C/C++ source code generated on  : 14-Jan-2018 07:15:05
 */

/*************************************************************************/
/* This automatically generated example C main file shows how to call    */
/* entry-point functions that MATLAB Coder generated. You must customize */
/* this file for your application. Do not modify this file directly.     */
/* Instead, make a copy of this file, modify it, and integrate it into   */
/* your development environment.                                         */
/*                                                                       */
/* This file initializes entry-point function arguments to a default     */
/* size and value before calling the entry-point functions. It does      */
/* not store or use any values returned from the entry-point functions.  */
/* If necessary, it does pre-allocate memory for returned values.        */
/* You can use this file as a starting point for a main function that    */
/* you can deploy in your application.                                   */
/*                                                                       */
/* After you copy the file, and before you deploy it, you must make the  */
/* following changes:                                                    */
/* * For variable-size function arguments, change the example sizes to   */
/* the sizes that your application requires.                             */
/* * Change the example values of function arguments to the values that  */
/* your application requires.                                            */
/* * If the entry-point functions return values, store these values or   */
/* otherwise use them as required by your application.                   */
/*                                                                       */
/*************************************************************************/
/* Include Files */
#include "rt_nonfinite.h"
#include "sobelEdge.h"
#include "main.h"
#include "sobelEdge_terminate.h"
#include "sobelEdge_initialize.h"

/* Function Declarations */
static void argInit_240x320_real32_T(real32_T result[76800]);
static real32_T argInit_real32_T(void);
static void main_sobelEdge(void);

/* Function Definitions */

/*
 * Arguments    : real32_T result[76800]
 * Return Type  : void
 */
static void argInit_240x320_real32_T(real32_T result[76800])
{
  int32_T idx0;
  int32_T idx1;

  /* Loop over the array to initialize each element. */
  for (idx0 = 0; idx0 < 240; idx0++) {
    for (idx1 = 0; idx1 < 320; idx1++) {
      /* Set the value of the array element.
         Change this value to the value that the application requires. */
      result[idx0 + 240 * idx1] = argInit_real32_T();
    }
  }
}

/*
 * Arguments    : void
 * Return Type  : real32_T
 */
static real32_T argInit_real32_T(void)
{
  return 0.0F;
}

/*
 * Arguments    : void
 * Return Type  : void
 */
static void main_sobelEdge(void)
{
  static real32_T magnitude[76800];
  static real32_T b[76800];

  /* Initialize function 'sobelEdge' input arguments. */
  /* Initialize function input argument 'Image'. */
  /* Call the entry-point 'sobelEdge'. */
  argInit_240x320_real32_T(b);
  sobelEdge(b, magnitude);
}

/*
 * Arguments    : int32_T argc
 *                const char * const argv[]
 * Return Type  : int32_T
 */
int32_T main(int32_T argc, const char * const argv[])
{
  (void)argc;
  (void)argv;

  /* Initialize the application.
     You do not need to do this more than one time. */
  sobelEdge_initialize();

  /* Invoke the entry-point functions.
     You can call entry-point functions multiple times. */
  main_sobelEdge();

  /* Terminate the application.
     You do not need to do this more than one time. */
  sobelEdge_terminate();
  return 0;
}

/*
 * File trailer for main.cu
 *
 * [EOF]
 */
