//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: sobelEdge_initialize.cu
//
// GPU Coder version                    : 1.0
// CUDA/C/C++ source code generated on  : 23-Jan-2018 10:11:24
//

// Include Files
#include "rt_nonfinite.h"
#include "sobelEdge.h"
#include "sobelEdge_initialize.h"

// Function Definitions

//
// Arguments    : void
// Return Type  : void
//
void sobelEdge_initialize()
{
  rt_InitInfAndNaN(8U);
}

//
// File trailer for sobelEdge_initialize.cu
//
// [EOF]
//
